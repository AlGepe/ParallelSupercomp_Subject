#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>

#define RADIUS        3
#define NUM_ELEMENTS  1000 

static void handleError(hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
#define cudaCheck( err ) (handleError(err, __FILE__, __LINE__ ))

__global__ void stencil_1d(int *in, int *out) {
	//PUT YOUR CODE HERE
	int array_length = sizeof(*in)/sizeof(in[0]);
	// Run whole array
	for(int i = 0; i < array_length; i++)
	{
		// Calculate for all neighbours and check
		for(int j = -RADIUS; j <= RADIUS; j++)
		{
			if(i + j < 0)
			{
				j += array_length;
			}
			else if (i+j > array_length)
			{
				j -= array_length;
			}
			out[i] += in[i+j];

		}

		void cpu_stencil_1d(int *in, int *out) {
			//PUT YOUR CODE HERE

		}

		int main() {
			//PUT YOUR CODE HERE - INPUT AND OUTPUT ARRAYS 
			float *input_array,
			      *output_array;

			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord( start, 0 );

			//PUT YOUR CODE HERE - DEVICE MEMORY ALLOCATION
			cudaCheck(hipMalloc((void**)&input_array, NUM_ELEMENTS*sizeof(float)));
			cudaCheck(hipMalloc((void**)&output_array, NUM_ELEMENTS*sizeof(float)));

			//PUT YOUR CODE HERE - KERNEL EXECUTION

			cudaCheck(hipPeekAtLastError());

			//PUT YOUR CODE HERE - COPY RESULT FROM DEVICE TO HOST

			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			float elapsedTime;
			hipEventElapsedTime( &elapsedTime, start, stop);
			printf("Total GPU execution time:  %3.1f ms\n", elapsedTime);
			hipEventDestroy(start);
			hipEventDestroy(stop);

			//PUT YOUR CODE HERE - FREE DEVICE MEMORY  
			cudaCheck(hipFree(input_array));

			struct timespec cpu_start, cpu_stop;
			clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_start);

			cpu_stencil_1d(in, out);

			clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_stop);
			double result = (cpu_stop.tv_sec - cpu_start.tv_sec) * 1e3 + (cpu_stop.tv_nsec - cpu_start.tv_nsec) / 1e6;
			printf( "CPU execution time:  %3.1f ms\n", result);

			return 0;
		}


